#include "hip/hip_runtime.h"
#include "CudaTools.h"

#include "hip/hip_runtime.h"
#include ""
#include "cuda_gl_interop.h"

#include "Terrain.h"


__device__ float noise(float x, float y)
{
	int n = int(x + (y + 62394) * 57);
	n = (n << 13) ^ n;
	return (1.0 - ((n * (n * n * 15731 + 789221) + 1376312589)
		& 0x7fffffff) / 1073741824.0);
}

__device__ float cosineInterpolate(float a, float b, float x)
{
	float ft = x * 3.1415927;
	float f = (1 - cos(ft)) * 0.5;

	return  a*(1 - f) + b*f;
}

__device__ float perlinHeight(float x, float y)
{
	float X = floor(x), Y = floor(y);

	float v1 = noise(X, Y);
	float v2 = noise(X + 1, Y);
	float v3 = noise(X, Y + 1);
	float v4 = noise(X + 1, Y + 1);

	return cosineInterpolate(
		cosineInterpolate(v1, v2, x - X),
		cosineInterpolate(v3, v4, x - X),
		y - Y
	);
}

__global__ void perlinNoise(float * d_heightmap, float f, float fd, float a, float ad, int iterations, int size)
{
	int ind = blockIdx.x * blockDim.x + threadIdx.x;
	if (ind < size*size)
	{
		float frequency = f;
		float amplitude = a;

		float x = ind % size, y = ind / size;

		float h = 0;

		for (int i = 0; i < iterations; i++)
		{
			h += perlinHeight(x / frequency, y / frequency)*amplitude;
			frequency /= fd;
			amplitude /= ad;

		}

		d_heightmap[ind] = h;
	}
}

__global__ void cuda_initArray(float * arr, float height, int size)
{
	int ind = blockIdx.x * blockDim.x + threadIdx.x;
	if (ind < size)
	{
		arr[ind] = height;
	}
}

__global__ void cuda_sqr(float * arr, int size)
{
	int ind = blockIdx.x * blockDim.x + threadIdx.x;
	if (ind < size*size)
	{
		arr[ind] = arr[ind] * arr[ind];
	}
}

__global__ void cuda_MapNormals(float * heightmap, float * normalmap, float h, int size)
{
	int ind = blockIdx.x * blockDim.x + threadIdx.x;
	if (ind < size*size)
	{
		float x = ind % size, y = ind / size;
		bool neighbours[4];
		neighbours[0] = y < size - 1;
		neighbours[1] = y > 0;
		neighbours[2] = x < size - 1;
		neighbours[3] = x > 0;

		float nh[4];

		if (neighbours[0])
			nh[0] = heightmap[ind + size]*h;
		else
			nh[0] = heightmap[ind] * h;
		if (neighbours[1])
			nh[1] = heightmap[ind - size] * h;
		else
			nh[1] = heightmap[ind] * h;
		if (neighbours[2])
			nh[2] = heightmap[ind + 1] * h;
		else
			nh[2] = heightmap[ind] * h;
		if (neighbours[3])
			nh[3] = heightmap[ind - 1] * h;
		else
			nh[3] = heightmap[ind] * h;

		vec3 n = normalize(vec3((nh[2] - nh[3]) / 2, 1, (nh[0] - nh[1]) / 2));

		normalmap[ind * 3] = n.x;
		normalmap[ind * 3 + 1] = n.y;
		normalmap[ind * 3 + 2] = n.z;
	}
}

__global__ void cuda_MapNormals(float * heightmap, float * watermap, float * normalmap, float h, int size)
{
	int ind = blockIdx.x * blockDim.x + threadIdx.x;
	if (ind < size*size)
	{
		float x = ind % size, y = ind / size;
		bool neighbours[4];
		neighbours[0] = y < size - 1;
		neighbours[1] = y > 0;
		neighbours[2] = x < size - 1;
		neighbours[3] = x > 0;

		float nh[4];

		if (neighbours[0])
			nh[0] = heightmap[ind + size] * h + watermap[ind + size];
		else
			nh[0] = heightmap[ind] * h + watermap[ind];
		if (neighbours[1])
			nh[1] = heightmap[ind - size] * h + watermap[ind - size];
		else
			nh[1] = heightmap[ind] * h + watermap[ind];
		if (neighbours[2])
			nh[2] = heightmap[ind + 1] * h + watermap[ind + 1];
		else
			nh[2] = heightmap[ind] * h + watermap[ind];
		if (neighbours[3])
			nh[3] = heightmap[ind - 1] * h + watermap[ind - 1];
		else
			nh[3] = heightmap[ind] * h + watermap[ind];

		vec3 n = normalize(vec3((nh[2] - nh[3]) / 2, 1, (nh[0] - nh[1]) / 2));

		normalmap[ind * 3] = n.x;
		normalmap[ind * 3 + 1] = n.y;
		normalmap[ind * 3 + 2] = n.z;
	}
}

__global__ void cuda_Add(float *Arr, float factor, int size)
{
	int ind = blockIdx.x * blockDim.x + threadIdx.x;
	if (ind < size*size)
	{
		Arr[ind] += factor;
		if (Arr[ind] < 0)
			Arr[ind] = 0;
	}
}

__global__ void cuda_elevate(float *heightmap, float offsetX, float offsetY, float centerX, float centerY, float outerRadius, float innerRadius, float factor, int innerSize, int size)
{
	int ind = blockIdx.x * blockDim.x + threadIdx.x;
	if (ind < innerSize*innerSize)
	{
		int x = ind % innerSize + offsetX;
		int y = ind / innerSize + offsetY;
		if (x < 0 || y < 0)
			return;
		if (x >= size || y >= size)
			return;

		float dist = sqrtf((centerX - x)*(centerX - x) + (centerY - y)*(centerY - y));
		if (dist > outerRadius)
			return;

		if (dist < innerRadius)
		{
			heightmap[x*size + y] += factor;
			return;
		}
		heightmap[x*size + y] += factor * cosineInterpolate(0, 1, (outerRadius - dist) / (outerRadius - innerRadius));
	}
}

__global__ void cuda_averagize(float *heightmap, float *heightBuffer, float offsetX, float offsetY, float centerX, float centerY, float outerRadius, float innerRadius, float factor, int innerSize, int size)
{
	int ind = blockIdx.x * blockDim.x + threadIdx.x;
	if (ind < innerSize*innerSize)
	{
		int x = ind % innerSize + offsetX;
		int y = ind / innerSize + offsetY;
		if (x < 0 || y < 0)
			return;
		if (x >= size || y >= size)
			return;

		int here = x*size + y;

		float dist = sqrtf((centerX - x)*(centerX - x) + (centerY - y)*(centerY - y));

		if (dist > outerRadius)
			return;

		float avg = (heightmap[here + 1] + heightmap[here - 1] + heightmap[here - size] + heightmap[here + size]) / 4;

		if (dist < innerRadius)
		{
			heightBuffer[here] = heightmap[here] * (1-factor) + avg * factor;
			return;
		}
		ind = x*size + y;

		if (outerRadius - innerRadius == 0)
			return;

		float fade = factor * cosineInterpolate(0, 1, (outerRadius - dist) / (outerRadius - innerRadius));

		heightBuffer[here] = heightmap[here] * (1-fade) + avg * fade;
	}
}

__global__ void cuda_plateao(float *heightmap, float height, float offsetX, float offsetY, float centerX, float centerY, float outerRadius, float innerRadius, float factor, bool above, bool below, int innerSize, int size)
{
	int ind = blockIdx.x * blockDim.x + threadIdx.x;
	if (ind < innerSize*innerSize)
	{
		unsigned int x = ind % innerSize + offsetX;
		int y = ind / innerSize + offsetY;
		if (x < 0 || y < 0)
			return;
		if (x >= size || y >= size)
			return;

		float dist = sqrtf((centerX - x)*(centerX - x) + (centerY - y)*(centerY - y));
		if (dist > outerRadius)
			return;

		float delta = factor * (height - heightmap[x*size + y]);

		if (delta < 0 && !above)
			return;
		if (delta > 0 && !below)
			return;

		if (dist < innerRadius)
		{
			heightmap[x*size + y] += delta;
			return;
		}

		if (outerRadius - innerRadius == 0)
			return;

		heightmap[x*size + y] += delta * cosineInterpolate(0, 1, (outerRadius - dist) / (outerRadius - innerRadius));
	}
}

__global__ void cuda_addBuffer(float *heightmap, float *heightBuffer, int size)
{
	int ind = blockIdx.x * blockDim.x + threadIdx.x;
	if (ind < size*size)
	{
		if(heightBuffer[ind] == 0) return;
		heightmap[ind] = heightBuffer[ind];
		heightBuffer[ind] = 0;
	}
}

__global__ void waterFlowI(float *heightmap, float * watermap, float * flow, float h, int size)
{
	int ind = blockIdx.x * blockDim.x + threadIdx.x;
	if (ind < size*size)
	{
		float x = ind % size, y = ind / size;
		bool neighbours[4] =
		{
			y < size - 1,
			y > 0,
			x < size - 1,
			x > 0
		};

		int n[4] = { ind + size, ind - size, ind + 1, ind - 1 };
		int f[4] = { ind * 4 ,ind * 4 + 1, ind * 4 + 2, ind * 4 + 3 };

		auto waterlvlDelta = [heightmap, watermap, ind, h](int nind)
		{ return (heightmap[ind]*h + watermap[ind]) - (heightmap[nind]*h + watermap[nind]); };


		for (int i = 0; i < 4; i++)
		{
			if (neighbours[i])
				flow[f[i]] = max(0.0f, flow[f[i]]*0.9f + waterlvlDelta(n[i])*0.5f);
			else
				flow[f[i]] = 0;
		}

		float totalFlow = flow[f[0]] + flow[f[1]] + flow[f[2]] + flow[f[3]];
		if (totalFlow == 0)
			return;

		float K = min(1.0f,	watermap[ind] / (totalFlow));

		for (int i = 0; i < 4; i++)
		{
			flow[f[i]] = K * flow[f[i]];
		}
	}
}

__global__ void waterFlowII(float* heightmap, float* normalmap, float* sediment, float * watermap, float * flow, float * velocity, int size)
{
	int ind = blockIdx.x * blockDim.x + threadIdx.x;
	if (ind < size*size)
	{
		float x = ind % size, y = ind / size;
		bool neighbours[4] =
		{
			y < size - 1,
			y > 0,
			x < size - 1,
			x > 0
		};

		int n[4] = { ind + size, ind - size, ind + 1, ind - 1 };
		int f[4] = { ind * 4 ,ind * 4 + 1, ind * 4 + 2, ind * 4 + 3 };
		int nf[4] = { n[0]*4 + 1, n[1]*4, n[2]*4 + 3, n[3]*4 + 2 };


		float outflow = flow[f[0]] + flow[f[1]] + flow[f[2]] + flow[f[3]];
		float inflow = 0.0f;

		for (int i = 0; i < 4; i++)
			if (neighbours[i])
				inflow += flow[nf[i]];

		float waterAvg = watermap[ind];

		watermap[ind] = watermap[ind] + (inflow - outflow)*0.5f;

		waterAvg += watermap[ind];
		waterAvg /= 2;
		
		if (waterAvg == 0)
		{
			velocity[ind * 2] = 0;
			velocity[ind * 2 + 1] = 0;
		}
		else
		{
			velocity[ind * 2] = (flow[f[0]] - flow[f[1]] + flow[nf[1]] - flow[nf[0]]) / 2 / waterAvg;
			velocity[ind * 2 + 1] = (flow[f[2]] - flow[f[3]] + flow[nf[3]] - flow[nf[2]]) / 2 / waterAvg;
		}

		/*if (abs(velocity[ind * 2]) > 1 || abs(velocity[ind * 2 + 1]) > 1)
		{
			heighttmap[ind] = 0;
		}*/

		float Kc = 0.01f;
		float Ke = 0.1f;
		float Kd = 1.0f;
		float C =
			Kc *
			min(0.3f, dot(vec3(0, 1, 0), vec3(normalmap[ind * 3], normalmap[ind * 3 + 1], normalmap[ind * 3 + 2]))) *
			//abs(outflow - inflow);
			sqrt(velocity[ind * 2]* velocity[ind * 2] + velocity[ind * 2 + 1]* velocity[ind * 2 + 1]);

		float delta = 0;

		if (sediment[ind] < 0)
			heightmap[ind] = 0;


		if (C > sediment[ind])
		{//erode
			delta = C - sediment[ind];
			heightmap[ind] -= delta * Ke;
			sediment[ind] += delta * Ke;
		}
		else
		{//deposit
			delta = sediment[ind] - C;
			heightmap[ind] += delta * Kd;
			sediment[ind] -= delta  * Kd;
		}

	}
}

__global__ void waterFlowIII(float * sediment, float * sedimentBuffer, float * velocity, int size)
{
	int ind = blockIdx.x * blockDim.x + threadIdx.x;
	if (ind < size*size)
	{
		float x = ind % size, y = ind / size;
		bool neighbours[4] =
		{
			y < size - 1,
			y > 0,
			x < size - 1,
			x > 0
		};

		int n[4] = { ind + size, ind - size, ind + 1, ind - 1 };
		

		sedimentBuffer[ind] = 0;
		sedimentBuffer[ind] = (1 - abs(velocity[ind * 2]))*(1 - abs(velocity[ind * 2 + 1])) * sediment[ind] * 0.5f;


		if (neighbours[0])
			if (velocity[n[0] * 2] < 0)
				sedimentBuffer[ind] += abs(velocity[n[0] * 2])*(1 - abs(velocity[n[0] * 2 + 1])) * sediment[n[0]] * 0.5f;
		if (neighbours[1])
			if (velocity[n[1] * 2] > 0)
				sedimentBuffer[ind] += abs(velocity[n[1] * 2])*(1 - abs(velocity[n[1] * 2 + 1])) * sediment[n[1]] * 0.5f;
		if (neighbours[2])
			if (velocity[n[2] * 2 + 1] < 0)
				sedimentBuffer[ind] += (1 - abs(velocity[n[2] * 2]))*abs(velocity[n[2] * 2 + 1]) * sediment[n[2]] * 0.5f;
		if (neighbours[3])
			if (velocity[n[3] * 2 + 1] > 0)
				sedimentBuffer[ind] += (1 - abs(velocity[n[3] * 2]))*abs(velocity[n[3] * 2 + 1]) * sediment[n[3]] * 0.5f;


	}
}

__global__ void waterFlowIV(float * sediment, float * sedimentBuffer, int size)
{
	int ind = blockIdx.x * blockDim.x + threadIdx.x;
	if (ind < size*size)
	{
		sediment[ind] = sedimentBuffer[ind];
		sedimentBuffer[ind] = 0;

	}
}

//__global__ void tempWeathering(float * heightMap, float *heightmapBuffer)

CudaTools::CudaTools(Terrain &terrain)
{
	cudaGLSetGLDevice(0);

	_terrain = &terrain;
	_size = terrain._size;
	N = _size*_size;
	M = 128;



	cudaGLRegisterBufferObject(terrain.vbo[1]);
	cudaGLRegisterBufferObject(terrain.vbo[2]);

	hipGraphicsGLRegisterBuffer(&cuda_vb_resources[0], terrain.vbo[1], cudaGraphicsMapFlagsNone); // heightmap
	hipGraphicsGLRegisterBuffer(&cuda_vb_resources[1], terrain.vbo[2], cudaGraphicsMapFlagsNone); // normals

	hipGraphicsGLRegisterBuffer(&cuda_vb_resources[2], terrain.vbo[3], cudaGraphicsMapFlagsNone); // w_heightmap
	hipGraphicsGLRegisterBuffer(&cuda_vb_resources[3], terrain.vbo[4], cudaGraphicsMapFlagsNone); // w_normals

	size_t s[4];
	s[0] = sizeof(GLfloat) * _size*_size;
	s[1] = sizeof(GLfloat) * _size*_size * 3;

	s[2] = sizeof(GLfloat) * _size*_size;
	s[3] = sizeof(GLfloat) * _size*_size * 3;


	hipGraphicsMapResources(4, cuda_vb_resources, 0);

	hipGraphicsResourceGetMappedPointer((void**)&d_heightmap, &s[0], cuda_vb_resources[0]);
	hipGraphicsResourceGetMappedPointer((void**)&d_normals, &s[1], cuda_vb_resources[1]);

	hipGraphicsResourceGetMappedPointer((void**)&d_watermap, &s[2], cuda_vb_resources[2]);
	hipGraphicsResourceGetMappedPointer((void**)&d_waterNormals, &s[3], cuda_vb_resources[3]);

	hipGraphicsUnmapResources(4, cuda_vb_resources, 0);

	hipMalloc((void**)&d_heightBuffer, sizeof(float) * _size * _size);
	hipMalloc((void**)&d_outflow, sizeof(float) * _size * _size * 4);
	hipMalloc((void**)&d_sediment, sizeof(float) * _size * _size);
	hipMalloc((void**)&d_velocity, sizeof(float) * _size * _size * 2);

	setHeight(0.0f, d_heightmap, _size*_size);
	setHeight(0.0f, d_watermap, _size*_size);
	setHeight(0.0f, d_sediment, _size*_size);

	cuda_initArray << <(N * 4 + M - 1) / M, M >> > (d_outflow, 0.0f, _size *_size * 4);
	cuda_initArray << <(N * 2 + M - 1) / M, M >> > (d_velocity, 0.0f, _size *_size * 2);

	fetchHeight();
	mapNormals();
}

CudaTools::~CudaTools()
{
	cudaGLUnregisterBufferObject(_terrain->vbo[1]);
	cudaGLUnregisterBufferObject(_terrain->vbo[2]);

	hipFree(d_heightBuffer);
	hipFree(d_outflow);
	hipFree(d_sediment);
	hipFree(d_velocity);
}

void CudaTools::setHeight(float height, float* arr, float size)
{
	cuda_initArray << <(N + M - 1) / M, M >> > (arr, height, size);

	hipDeviceSynchronize();
}

void CudaTools::square()
{
	cuda_sqr << <(N + M - 1) / M, M >> > ((float*)d_heightmap, _size);

	hipDeviceSynchronize();
}

void CudaTools::PerlinNoise(float frequency, float frequencyDivider, float amplitude, float amplitudeDivider, int iterations)
{
	perlinNoise << <(N + M - 1) / M, M >> >(d_heightmap, frequency, frequencyDivider, amplitude, amplitudeDivider, iterations, _size);

	hipDeviceSynchronize();
}

void CudaTools::mapNormals()
{
	cuda_MapNormals << < (N + M - 1) / M, M >> >(d_heightmap, d_normals, h, _size);
	hipDeviceSynchronize();

	cuda_MapNormals << < (N + M - 1) / M, M >> >(d_heightmap, d_watermap, d_waterNormals, h, _size);
	hipDeviceSynchronize();

}

void CudaTools::fetchHeight()
{
	hipMemcpy(_terrain->heightmap, d_heightmap, _size *_size * sizeof(float), hipMemcpyDeviceToHost);
}

void CudaTools::elevate(const vec2 &position, float outerRadius, float innerRadius, float factor)
{
	int _x = floor(position.x - outerRadius);
	int _y = floor(position.y - outerRadius);

	int size = ceil(2 * outerRadius) + 1;

	int n = size*size;

	cuda_elevate << <(n + M - 1) / M, M >> >(d_heightmap, _x, _y, position.x, position.y, outerRadius, innerRadius, factor, size, _size);
	hipDeviceSynchronize();
}

void CudaTools::averagize(const vec2 &position, float outerRadius, float innerRadius, float factor)
{
	int _x = floor(position.x - outerRadius);
	int _y = floor(position.y - outerRadius);

	int size = ceil(2 * outerRadius) + 1;

	int n = size*size;
	cuda_initArray << <(N + M - 1) / M, M >> > (d_heightBuffer, 0, _size);
	hipDeviceSynchronize();
	cuda_averagize << <(n + M - 1) / M, M >> >(d_heightmap, d_heightBuffer, _x, _y, position.x, position.y, outerRadius, innerRadius, factor, size, _size);
	hipDeviceSynchronize();
	cuda_addBuffer << <(N + M - 1) / M, M >> >(d_heightmap, d_heightBuffer, _size);
	hipDeviceSynchronize();
}

void CudaTools::plateau(const vec3 &position, float outerRadius, float innerRadius, float factor, bool above, bool below)
{
	if (!(above || below))
		return;

	int _x = floor(position.x - outerRadius);
	int _y = floor(position.y - outerRadius);

	int size = ceil(2 * outerRadius) + 1;

	int n = size*size;

	cuda_plateao << <(n + M - 1) / M, M >> >(d_heightmap, position.z, _x, _y, position.x, position.y, outerRadius, innerRadius, factor, above, below, size, _size);
	hipDeviceSynchronize();
}

void CudaTools::waterFlow(float H)
{
	waterFlowI << <(N + M - 1) / M, M >> >(d_heightmap, d_watermap, d_outflow, h, _size);
	hipDeviceSynchronize();
	waterFlowII << <(N + M - 1) / M, M >> >(d_heightmap, d_normals, d_sediment, d_watermap, d_outflow, d_velocity, _size);
	hipDeviceSynchronize();
	waterFlowIII << <(N + M - 1) / M, M >> >(d_sediment, d_heightBuffer, d_velocity, _size);
	hipDeviceSynchronize();
	waterFlowIV << <(N + M - 1) / M, M >> > (d_sediment, d_heightBuffer, _size);

	hipDeviceSynchronize();
}

void CudaTools::addWater(float factor)
{
	cuda_Add << <(N + M - 1) / M, M >> >(d_watermap, factor, _size);
}

void CudaTools::raindrop(const vec2 & position, float outerRadius, float innerRadius, float factor)
{
	int _x = floor(position.x - outerRadius);
	int _y = floor(position.y - outerRadius);

	int size = ceil(2 * outerRadius) + 1;

	int n = size*size;

	cuda_elevate << <(n + M - 1) / M, M >> >(d_watermap, _x, _y, position.x, position.y, outerRadius, innerRadius, factor, size, _size);
	hipDeviceSynchronize();
}


