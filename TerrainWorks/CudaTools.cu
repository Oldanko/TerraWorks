#include "hip/hip_runtime.h"
#include "CudaTools.h"

#include "hip/hip_runtime.h"
#include ""
#include "cuda_gl_interop.h"

#include "Terrain.h"

__device__ float noise(float x, float y)
{
	int n = int(x + (y + 62394) * 57);
	n = (n << 13) ^ n;
	return (1.0 - ((n * (n * n * 15731 + 789221) + 1376312589)
		& 0x7fffffff) / 1073741824.0);
}

__device__ float cosineInterpolate(float a, float b, float x)
{
	float ft = x * 3.1415927;
	float f = (1 - cos(ft)) * 0.5;

	return  a*(1 - f) + b*f;
}

__device__ float perlinHeight(float x, float y)
{
	float X = floor(x), Y = floor(y);

	float v1 = noise(X, Y);
	float v2 = noise(X + 1, Y);
	float v3 = noise(X, Y + 1);
	float v4 = noise(X + 1, Y + 1);

	return cosineInterpolate(
		cosineInterpolate(v1, v2, x - X),
		cosineInterpolate(v3, v4, x - X),
		y - Y
	);
}

__global__ void perlinNoise(float * d_heightmap, float f, float fd, float a, float ad, int iterations, int size)
{
	int ind = blockIdx.x * blockDim.x + threadIdx.x;
	if (ind < size*size)
	{
		float frequency = f;
		float amplitude = a;

		float x = ind % size, y = ind / size;

		float h = 0;

		for (int i = 0; i < iterations; i++)
		{
			h += perlinHeight(x / frequency, y / frequency)*amplitude;
			frequency /= fd;
			amplitude /= ad;

		}

		d_heightmap[ind] = h;
	}
}

__global__ void cuda_initArray(float * arr, float height, int size)
{
	int ind = blockIdx.x * blockDim.x + threadIdx.x;
	if (ind < size*size)
	{
		arr[ind] = height;
	}
}

__global__ void cuda_sqr(float * arr, int size)
{
	int ind = blockIdx.x * blockDim.x + threadIdx.x;
	if (ind < size*size)
	{
		arr[ind] = arr[ind] * arr[ind];
	}
}

__global__ void cuda_MapNormals(float * heightmap, float * normalmap, int size)
{
	int ind = blockIdx.x * blockDim.x + threadIdx.x;
	if (ind < size*size)
	{
		float x = ind % size, y = ind / size;
		bool neighbours[4];
		neighbours[0] = y < size - 1;
		neighbours[1] = y > 0;
		neighbours[2] = x < size - 1;
		neighbours[3] = x > 0;

		float nh[4];

		if (neighbours[0])
			nh[0] = heightmap[ind + size];
		else
			nh[0] = heightmap[ind];
		if (neighbours[1])
			nh[1] = heightmap[ind - size];
		else
			nh[1] = heightmap[ind];
		if (neighbours[2])
			nh[2] = heightmap[ind + 1];
		else
			nh[2] = heightmap[ind];
		if (neighbours[3])
			nh[3] = heightmap[ind - 1];
		else
			nh[3] = heightmap[ind];

		vec3 n = normalize(vec3((nh[2] - nh[3]) / 2, 1, (nh[0] - nh[1]) / 2));

		normalmap[ind * 3] = n.x;
		normalmap[ind * 3 + 1] = n.y;
		normalmap[ind * 3 + 2] = n.z;
	}
}

__global__ void cuda_MapNormals(float * heightmap, float * watermap, float * normalmap, int size)
{
	int ind = blockIdx.x * blockDim.x + threadIdx.x;
	if (ind < size*size)
	{
		float x = ind % size, y = ind / size;
		bool neighbours[4];
		neighbours[0] = y < size - 1;
		neighbours[1] = y > 0;
		neighbours[2] = x < size - 1;
		neighbours[3] = x > 0;

		float nh[4];

		if (neighbours[0])
			nh[0] = heightmap[ind + size] + watermap[ind + size];
		else
			nh[0] = heightmap[ind] + watermap[ind];
		if (neighbours[1])
			nh[1] = heightmap[ind - size] + watermap[ind - size];
		else
			nh[1] = heightmap[ind] + watermap[ind];
		if (neighbours[2])
			nh[2] = heightmap[ind + 1] + watermap[ind + 1];
		else
			nh[2] = heightmap[ind] + watermap[ind];
		if (neighbours[3])
			nh[3] = heightmap[ind - 1] + watermap[ind - 1];
		else
			nh[3] = heightmap[ind] + watermap[ind];

		vec3 n = normalize(vec3((nh[2] - nh[3]) / 2, 1, (nh[0] - nh[1]) / 2));

		normalmap[ind * 3] = n.x;
		normalmap[ind * 3 + 1] = n.y;
		normalmap[ind * 3 + 2] = n.z;
	}
}


__global__ void cuda_elevate(float *heightmap, float offsetX, float offsetY, float centerX, float centerY, float outerRadius, float innerRadius, float factor, int innerSize, int size)
{
	int ind = blockIdx.x * blockDim.x + threadIdx.x;
	if (ind < innerSize*innerSize)
	{
		int x = ind % innerSize + offsetX;
		int y = ind / innerSize + offsetY;
		if (x < 0 || y < 0)
			return;
		if (x >= size || y >= size)
			return;

		float dist = sqrtf((centerX - x)*(centerX - x) + (centerY - y)*(centerY - y));
		if (dist > outerRadius)
			return;

		if (dist < innerRadius)
		{
			heightmap[x*size + y] += factor;
			return;
		}
		heightmap[x*size + y] += factor * cosineInterpolate(0, 1, (outerRadius - dist) / (outerRadius - innerRadius));
	}
}

__global__ void cuda_averagize(float *heightmap, float *heightBuffer, float offsetX, float offsetY, float centerX, float centerY, float outerRadius, float innerRadius, float factor, int innerSize, int size)
{
	int ind = blockIdx.x * blockDim.x + threadIdx.x;
	if (ind < innerSize*innerSize)
	{
		int x = ind % innerSize + offsetX;
		int y = ind / innerSize + offsetY;
		if (x < 0 || y < 0)
			return;
		if (x >= size || y >= size)
			return;

		int here = x*size + y;

		float dist = sqrtf((centerX - x)*(centerX - x) + (centerY - y)*(centerY - y));

		if (dist > outerRadius)
			return;

		float avg = (heightmap[here + 1] + heightmap[here - 1] + heightmap[here - size] + heightmap[here + size]) / 4;

		if (dist < innerRadius)
		{
			heightBuffer[here] = heightmap[here] * (1-factor) + avg * factor;
			return;
		}
		ind = x*size + y;

		if (outerRadius - innerRadius == 0)
			return;

		float fade = factor * cosineInterpolate(0, 1, (outerRadius - dist) / (outerRadius - innerRadius));

		heightBuffer[here] = heightmap[here] * (1-fade) + avg * fade;
	}
}

__global__ void cuda_plateao(float *heightmap, float height, float offsetX, float offsetY, float centerX, float centerY, float outerRadius, float innerRadius, float factor, bool above, bool below, int innerSize, int size)
{
	int ind = blockIdx.x * blockDim.x + threadIdx.x;
	if (ind < innerSize*innerSize)
	{
		unsigned int x = ind % innerSize + offsetX;
		int y = ind / innerSize + offsetY;
		if (x < 0 || y < 0)
			return;
		if (x >= size || y >= size)
			return;

		float dist = sqrtf((centerX - x)*(centerX - x) + (centerY - y)*(centerY - y));
		if (dist > outerRadius)
			return;

		float delta = factor * (height - heightmap[x*size + y]);

		if (delta < 0 && !above)
			return;
		if (delta > 0 && !below)
			return;

		if (dist < innerRadius)
		{
			heightmap[x*size + y] += delta;
			return;
		}

		if (outerRadius - innerRadius == 0)
			return;

		heightmap[x*size + y] += delta * cosineInterpolate(0, 1, (outerRadius - dist) / (outerRadius - innerRadius));
	}
}

__global__ void cuda_addBuffer(float *heightmap, float *heightBuffer, int size)
{
	int ind = blockIdx.x * blockDim.x + threadIdx.x;
	if (ind < size*size)
	{
		if(heightBuffer[ind] == 0) return;
		heightmap[ind] = heightBuffer[ind];
		heightBuffer[ind] = 0;
	}
}


CudaTools::CudaTools(Terrain &terrain)
{
	cudaGLSetGLDevice(0);

	_terrain = &terrain;
	_size = terrain._size;
	N = _size*_size;
	M = 128;



	cudaGLRegisterBufferObject(terrain.vbo[1]);
	cudaGLRegisterBufferObject(terrain.vbo[2]);

	hipGraphicsGLRegisterBuffer(&cuda_vb_resources[0], terrain.vbo[1], cudaGraphicsMapFlagsNone); // heightmap
	hipGraphicsGLRegisterBuffer(&cuda_vb_resources[1], terrain.vbo[2], cudaGraphicsMapFlagsNone); // normals

	hipGraphicsGLRegisterBuffer(&cuda_vb_resources[2], terrain.vbo[3], cudaGraphicsMapFlagsNone); // w_heightmap
	hipGraphicsGLRegisterBuffer(&cuda_vb_resources[3], terrain.vbo[4], cudaGraphicsMapFlagsNone); // w_normals

	size_t s[4];
	s[0] = sizeof(GLfloat) * _size*_size;
	s[1] = sizeof(GLfloat) * _size*_size * 3;

	s[2] = sizeof(GLfloat) * _size*_size;
	s[3] = sizeof(GLfloat) * _size*_size * 3;


	hipGraphicsMapResources(4, cuda_vb_resources, 0);

	hipGraphicsResourceGetMappedPointer((void**)&d_heightmap, &s[0], cuda_vb_resources[0]);
	hipGraphicsResourceGetMappedPointer((void**)&d_normals, &s[1], cuda_vb_resources[1]);

	hipGraphicsResourceGetMappedPointer((void**)&d_watermap, &s[2], cuda_vb_resources[2]);
	hipGraphicsResourceGetMappedPointer((void**)&d_waterNormals, &s[3], cuda_vb_resources[3]);

	hipGraphicsUnmapResources(4, cuda_vb_resources, 0);

	hipMalloc((void**)&d_heightBuffer, sizeof(float) * _size * _size);
	hipMalloc((void**)&d_waterBuffer, sizeof(float) * _size * _size);
	hipMalloc((void**)&d_sediment, sizeof(float) * _size * _size);

	setHeight(0.0f, d_heightmap);
	setHeight(10.0f, d_watermap);
	fetchHeight();
	mapNormals();
}

CudaTools::~CudaTools()
{
	cudaGLUnregisterBufferObject(_terrain->vbo[1]);
	cudaGLUnregisterBufferObject(_terrain->vbo[2]);

	hipFree(d_heightBuffer);
}

void CudaTools::setHeight(float height, float* arr)
{
	cuda_initArray << <(N + M - 1) / M, M >> > ((float*)arr, height, _size);

	hipDeviceSynchronize();
}

void CudaTools::square()
{
	cuda_sqr << <(N + M - 1) / M, M >> > ((float*)d_heightmap, _size);

	hipDeviceSynchronize();
}

void CudaTools::PerlinNoise(float frequency, float frequencyDivider, float amplitude, float amplitudeDivider, int iterations)
{
	perlinNoise << <(N + M - 1) / M, M >> >(d_heightmap, frequency, frequencyDivider, amplitude, amplitudeDivider, iterations, _size);

	hipDeviceSynchronize();
}

void CudaTools::mapNormals()
{
	cuda_MapNormals << < (N + M - 1) / M, M >> >(d_heightmap, d_normals, _size);
	hipDeviceSynchronize();

	cuda_MapNormals << < (N + M - 1) / M, M >> >(d_heightmap, d_watermap, d_waterNormals, _size);
	hipDeviceSynchronize();

}

void CudaTools::fetchHeight()
{
	hipMemcpy(_terrain->heightmap, d_heightmap, _size *_size * sizeof(float), hipMemcpyDeviceToHost);
}

void CudaTools::elevate(const vec2 &position, float outerRadius, float innerRadius, float factor)
{
	int _x = floor(position.x - outerRadius);
	int _y = floor(position.y - outerRadius);

	int size = ceil(2 * outerRadius) + 1;

	int n = size*size;

	cuda_elevate << <(n + M - 1) / M, M >> >(d_heightmap, _x, _y, position.x, position.y, outerRadius, innerRadius, factor, size, _size);
	hipDeviceSynchronize();
}

void CudaTools::averagize(const vec2 &position, float outerRadius, float innerRadius, float factor)
{
	int _x = floor(position.x - outerRadius);
	int _y = floor(position.y - outerRadius);

	int size = ceil(2 * outerRadius) + 1;

	int n = size*size;
	cuda_initArray << <(N + M - 1) / M, M >> > (d_heightBuffer, 0, _size);
	hipDeviceSynchronize();
	cuda_averagize << <(n + M - 1) / M, M >> >(d_heightmap, d_heightBuffer, _x, _y, position.x, position.y, outerRadius, innerRadius, factor, size, _size);
	hipDeviceSynchronize();
	cuda_addBuffer << <(N + M - 1) / M, M >> >(d_heightmap, d_heightBuffer, _size);
	hipDeviceSynchronize();
}

void CudaTools::plateau(const vec3 &position, float outerRadius, float innerRadius, float factor, bool above, bool below)
{
	if (!(above || below))
		return;

	int _x = floor(position.x - outerRadius);
	int _y = floor(position.y - outerRadius);

	int size = ceil(2 * outerRadius) + 1;

	int n = size*size;

	cuda_plateao << <(n + M - 1) / M, M >> >(d_heightmap, position.z, _x, _y, position.x, position.y, outerRadius, innerRadius, factor, above, below, size, _size);
	hipDeviceSynchronize();
}


